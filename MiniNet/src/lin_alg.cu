#include <iostream>
#include <hip/hip_runtime.h>

// Simple CUDA kernel to add two arrays
__global__ void add_arrays(int* A, int* B, int* C, int N) {
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    if (idx < N) {
        C[idx] = A[idx] + B[idx];
    }
}

int main() {
    int N = 1000;  // Number of elements in the arrays
    int size = N * sizeof(int);

    // Host arrays
    int* h_A = (int*)malloc(size);
    int* h_B = (int*)malloc(size);
    int* h_C = (int*)malloc(size);

    // Initialize arrays
    for (int i = 0; i < N; ++i) {
        h_A[i] = i;
        h_B[i] = i * 2;
    }

    // Device arrays
    int *d_A, *d_B, *d_C;
    hipMalloc(&d_A, size);
    hipMalloc(&d_B, size);
    hipMalloc(&d_C, size);

    // Copy data from host to device
    hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice);

    // Launch kernel with 1 block and N threads
    add_arrays<<<(N + 255) / 256, 256>>>(d_A, d_B, d_C, N);

    // Copy result from device to host
    hipMemcpy(h_C, d_C, size, hipMemcpyDeviceToHost);

    // Verify the result
    for (int i = 0; i < N; ++i) {
        if (h_C[i] != h_A[i] + h_B[i]) {
            std::cerr << "Error at index " << i << ": " << h_C[i] << " != " << h_A[i] + h_B[i] << std::endl;
            return -1;
        }
    }

    std::cout << "Test passed!" << std::endl;

    // Free memory
    free(h_A);
    free(h_B);
    free(h_C);
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);

    return 0;
}
